#include <iostream>  
#include <hip/hip_runtime.h>

using namespace std;

// CUDA kernel to add two vectors
__global__ void addVectors(int* A, int* B, int* C, int n) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() 
{
    int n = 1000000; // Size of vectors
    int size = n * sizeof(int);

    int *A, *B, *C; // Host pointers
    int *dev_A, *dev_B, *dev_C; // Device pointers

    // Allocate pinned memory on the host for faster transfers
    hipHostMalloc(&A, size, hipHostMallocDefault);  
    hipHostMalloc(&B, size, hipHostMallocDefault);  
    hipHostMalloc(&C, size, hipHostMallocDefault);

    // Initialize host vectors
    for (int i = 0; i < n; i++) {
        A[i] = i;
        B[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc(&dev_A, size);  
    hipMalloc(&dev_B, size);  
    hipMalloc(&dev_C, size);

    // Create CUDA events to time the kernel execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); // Start timing

    // Copy data from host to device
    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);  
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    // Define execution configuration
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    addVectors<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C, n);

    // Copy the result back to the host
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    // Stop timing and calculate elapsed time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print a few sample results
    cout << "Sample output (first 10 elements): ";
    for (int i = 0; i < 10; i++) {
        cout << C[i] << " ";
    }
    cout << endl;

    // Print total execution time in seconds
    cout << "Total kernel execution time: " << milliseconds / 1000 << " seconds" << endl;

    // Free device and host memory
    hipFree(dev_A);  
    hipFree(dev_B);  
    hipFree(dev_C);  
    hipHostFree(A);  
    hipHostFree(B);  
    hipHostFree(C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

/*
To compile and run:

1. Compile: nvcc vect_add.cu 
2. Run:     ./a.out
*/